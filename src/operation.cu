
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstddef>


const std::size_t THREADS_PER_1D_BLOCK = 1024;
const std::size_t THREADS_PER_2D_BLOCK = 32;


__global__ void copy_(unsigned int size, const double* arrayIn, double* arrayOut) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    arrayOut[index] = arrayIn[index];
}


/** Initializes an array with zeros. */
__global__ void zero_(unsigned int size, double* array) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    array[index] = 0;
}

void zero(std::size_t size, double* array) {
    std::size_t gridSize = (size + THREADS_PER_1D_BLOCK - 1) / THREADS_PER_1D_BLOCK;
    zero_<<<gridSize, THREADS_PER_1D_BLOCK>>>(size, array);
}


/** Vector(Matrix) Addition */
__global__ void add_(unsigned int size, const double* vector1, const double* vector2, double* vectorOut) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    vectorOut[index] = vector1[index] + vector2[index];
}

void add(std::size_t size, const double* vector1, const double* vector2, double* vectorOut) {
    std::size_t gridSize = (size + THREADS_PER_1D_BLOCK - 1) / THREADS_PER_1D_BLOCK;
    add_<<<gridSize, THREADS_PER_1D_BLOCK>>>(size, vector1, vector2, vectorOut);
}


/** Vector(Matrix) Subtraction */
__global__ void subtract_(unsigned int size, const double* vector1, const double* vector2, double* vectorOut) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    vectorOut[index] = vector1[index] - vector2[index];
}

void subtract(std::size_t size, const double* vector1, const double* vector2, double* vectorOut) {
    std::size_t gridSize = (size + THREADS_PER_1D_BLOCK - 1) / THREADS_PER_1D_BLOCK;
    subtract_<<<gridSize, THREADS_PER_1D_BLOCK>>>(size, vector1, vector2, vectorOut);
}


/** Vector(Matrix) Multiplication (Hadamard Product) */
__global__ void multiply_(unsigned int size, const double* vector1, const double* vector2, double* vectorOut) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    vectorOut[index] = vector1[index] * vector2[index];
}

void multiply(std::size_t size, const double* vector1, const double* vector2, double* vectorOut) {
    std::size_t gridSize = (size + THREADS_PER_1D_BLOCK - 1) / THREADS_PER_1D_BLOCK;
    multiply_<<<gridSize, THREADS_PER_1D_BLOCK>>>(size, vector1, vector2, vectorOut);
}


/** Dot Product */
__global__ void dotVectorsWithMatrixOut_(
    unsigned int vector1Size, unsigned int vector2Size,
    const double* vector1, const double* vector2, double* matrixOut
) {
    const unsigned int index1 = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int index2 = blockIdx.y * blockDim.y + threadIdx.y;
    if (index1 >= vector1Size || index2 >= vector2Size) return;
    matrixOut[index1 * vector2Size + index2] = vector1[index1] * vector2[index2];
}

void dotVectorsWithMatrixOut(
    std::size_t vector1Size, std::size_t vector2Size,
    const double* vector1, const double* vector2, double* matrixOut
) {
    dim3 gridDim((vector1Size + THREADS_PER_2D_BLOCK - 1) / THREADS_PER_2D_BLOCK,
                 (vector2Size + THREADS_PER_2D_BLOCK - 1) / THREADS_PER_2D_BLOCK);
    dim3 blockDim(THREADS_PER_2D_BLOCK, THREADS_PER_2D_BLOCK);
    dotVectorsWithMatrixOut_<<<gridDim, blockDim>>>(vector1Size, vector2Size, vector1, vector2, matrixOut);
}


__global__ void dotMatrixVectorMultiply_(
    unsigned int numOfMatrixRows, unsigned int numOfMatrixCols,
    const double* matrix, const double* vector, double* matrixOut
) {
    const unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row >= numOfMatrixRows || col >= numOfMatrixCols) return;
    matrixOut[row * numOfMatrixCols + col] = matrix[row * numOfMatrixCols + col] * vector[col];
}

__global__ void dotMatrixVectorSumReduction_(unsigned int numOfMatrixRows, unsigned int numOfMatrixCols, double* matrix) {
    const unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int sharedDataIndex = threadIdx.x * blockDim.y + threadIdx.y;
    if (row >= numOfMatrixRows) return;

    extern __shared__ double sharedData[];
    sharedData[sharedDataIndex] = (col < numOfMatrixCols) ? matrix[row * numOfMatrixCols + col] : 0;
    __syncthreads();

    for (unsigned int stride = blockDim.y / 2; stride > 0; stride >>= 1) {
        if (threadIdx.y < stride)
            sharedData[sharedDataIndex] += sharedData[sharedDataIndex + stride];
        __syncthreads();
    }

    if (threadIdx.y == 0) matrix[row * gridDim.y + blockIdx.y] = sharedData[sharedDataIndex];
}

void dotMatrixVector(
    std::size_t numOfMatrixRows, std::size_t numOfMatrixCols, const double* matrix,
    const double* vector, double* helperMatrix, double* vectorOut
) {
    dim3 gridDim((numOfMatrixRows + THREADS_PER_2D_BLOCK - 1) / THREADS_PER_2D_BLOCK,
                 (numOfMatrixCols + THREADS_PER_2D_BLOCK - 1) / THREADS_PER_2D_BLOCK);
    dim3 blockDim(THREADS_PER_2D_BLOCK, THREADS_PER_2D_BLOCK);
    std::size_t sharedMemSize = blockDim.x * blockDim.y * sizeof(double);

    dotMatrixVectorMultiply_<<<gridDim, blockDim>>>(numOfMatrixRows, numOfMatrixCols, matrix, vector, helperMatrix);

    while (numOfMatrixCols > 1) {
        dotMatrixVectorSumReduction_<<<gridDim, blockDim, sharedMemSize>>>(numOfMatrixRows, numOfMatrixCols, helperMatrix);
        numOfMatrixCols = gridDim.y;
        gridDim.y = (numOfMatrixCols + THREADS_PER_2D_BLOCK - 1) / THREADS_PER_2D_BLOCK;
    }

    std::size_t gridSize = (numOfMatrixRows + THREADS_PER_1D_BLOCK - 1) / THREADS_PER_1D_BLOCK;
    copy_<<<gridSize, THREADS_PER_1D_BLOCK>>>(numOfMatrixRows, helperMatrix, vectorOut);
}


/** Dot Product */
__global__ void dotVectorMatrix_(
    unsigned int numOfMatrixRows, unsigned int numOfMatrixCols,
    const double* vector, const double* matrix, double* vectorOut
) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numOfMatrixCols) return;
    vectorOut[index] = 0;  // initialize all elements to 0
    for (unsigned int i = 0; i < numOfMatrixRows; ++i)
        vectorOut[index] += matrix[i * numOfMatrixCols + index] * vector[i];
}

void dotVectorMatrix(
    std::size_t numOfMatrixRows, std::size_t numOfMatrixCols,
    const double* vector, const double* matrix, double* vectorOut
) {
    std::size_t gridSize = (numOfMatrixCols + THREADS_PER_1D_BLOCK - 1) / THREADS_PER_1D_BLOCK;
    dotVectorMatrix_<<<gridSize, THREADS_PER_1D_BLOCK>>>(numOfMatrixRows, numOfMatrixCols, vector, matrix, vectorOut);
}


/** Sigmoid Function: sigmoid(z) = 1 / (1 + (e ^ -z)) */
__global__ void sigmoid_(unsigned int size, const double* z, double* functionOut) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    functionOut[index] = 1 / (1 + std::exp(-z[index]));
}

void sigmoid(std::size_t size, const double* z, double* functionOut) {
    std::size_t gridSize = (size + THREADS_PER_1D_BLOCK - 1) / THREADS_PER_1D_BLOCK;
    sigmoid_<<<gridSize, THREADS_PER_1D_BLOCK>>>(size, z, functionOut);
}


/** Derivative of Sigmoid Function: sigmoid'(z) = sigmoid(z) * (1 - sigmoid(z)) */
__global__ void sigmoidPrime_(unsigned int size, const double* z, double* functionOut) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    double sigmoidZ = 1 / (1 + std::exp(-z[index]));
    functionOut[index] = sigmoidZ * (1 - sigmoidZ);
}

void sigmoidPrime(std::size_t size, const double* z, double* functionOut) {
    std::size_t gridSize = (size + THREADS_PER_1D_BLOCK - 1) / THREADS_PER_1D_BLOCK;
    sigmoidPrime_<<<gridSize, THREADS_PER_1D_BLOCK>>>(size, z, functionOut);
}


/** Reduces the cost with specified learning rate. */
__global__ void reduceCost_(
    unsigned int size, double learningRate,
    double miniBatchSize, const double* nabla, double* out
) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;
    out[index] -= (learningRate / miniBatchSize) * nabla[index];
}

void reduceCost(
    std::size_t size, double learningRate,
    double miniBatchSize, const double* nabla, double* out
) {
    std::size_t gridSize = (size + THREADS_PER_1D_BLOCK - 1) / THREADS_PER_1D_BLOCK;
    reduceCost_<<<gridSize, THREADS_PER_1D_BLOCK>>>(size, learningRate, miniBatchSize, nabla, out);
}